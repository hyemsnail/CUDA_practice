
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA 커널 함수
__global__ void kernel() {
    printf("Hello, World!\n");
}

int main(void) {
    kernel<<<1,1>>>();  // GPU에서 커널 실행
    hipDeviceSynchronize();  // GPU 연산 완료 대기
    fflush(stdout);  // 출력 버퍼 비우기
    return 0;
}
